#include "hip/hip_runtime.h"
//
#include "fft.h"
//
unordered_map<string, DataPoint> fingerPrint(long *audio, long sampleLength)
{
	//long sampleLength; //Read the length of the sample
	hipError_t cudaStatus;

	//Load wav file into memory
	//long *audio = readData(&sampleLength);
	int numChunks = sampleLength / CHUNK_SIZE;

	//Host memory
	hipfftComplex **wavData = new hipfftComplex* [numChunks];
	hipfftComplex **fftData = new hipfftComplex* [numChunks];

	//Creating Complex numbers from our readings
	for (int times = 0; times < numChunks; times++)
	{
		hipfftComplex *complex = new hipfftComplex[CHUNK_SIZE];
		fftData[times] = new hipfftComplex[CHUNK_SIZE];
		for (int i = 0; i < CHUNK_SIZE; i++)
		{
			complex[i].x = audio[times * CHUNK_SIZE + i];
			complex[i].y = 0;
		}
		wavData[times] = complex;
	}

	hipfftComplex *d_wavData;
	hipfftComplex *d_fftData;

	//The actual transform
	hipfftHandle plan;
	hipfftPlan1d(&plan, CHUNK_SIZE, HIPFFT_C2C, 1);

	//FFT
	for (int i = 0; i < numChunks; i++)
	{
		cudaStatus = hipMalloc(&d_wavData, CHUNK_SIZE  * sizeof(hipfftComplex));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "d_wav malloc failed");
		}

		cudaStatus = hipMalloc(&d_fftData, CHUNK_SIZE * sizeof(hipfftComplex));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "d_fft malloc failed");
		}

		//Copy over wav data
		cudaStatus = hipMemcpy(d_wavData, wavData[i], CHUNK_SIZE * sizeof(hipfftComplex), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "d_wavData Memcpy failed");
		}

		hipfftExecC2C(plan, (hipfftComplex*)d_wavData, (hipfftComplex*) d_fftData, 1);

		//Copy fft data back
		cudaStatus = hipMemcpy(fftData[i], d_fftData, CHUNK_SIZE * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "fftData Memcpy failed\n");
		}
	}
	
	delete wavData;
	hipFree(d_wavData);
	hipFree(d_fftData);
	hipfftDestroy(plan);
	hipDeviceReset();

	cout << "FFT Complete." << endl;

	//Create an array of hash points
	double **highScores = new double * [numChunks];
	for (int i = 0; i < numChunks; i ++)
	{
		highScores[i] = new double[5];
		for (int j = 0; j < 5; j++)
		{
			highScores[i][j] = 0;
		}
		
	}

	//Scale the data down
	for (int t = 0; t < numChunks; t++)
	{
		for (int freq = LOWER_LIMIT; freq < UPPER_LIMIT - 1; freq ++)
		{
			double magnitude = log(abs(fftData[t][freq].x)) + 1;

			int index = getIndex(freq);

			if (magnitude > highScores[t][index])
			{
				highScores[t][index] = magnitude;
			}
		}
	}
	cout << "Reduction Complete." << endl;

	//Store the matches
	unordered_map<string, DataPoint> storage;
	int fuzz = 2; // fuzziness
	for (int t = 0; t < numChunks; t++)
	{
		string hash = "";
		for (int j = 0; j < 5; j++)
		{
			//concat our hash with a fuzziness
			hash += to_string((highScores[t][j] - ((int)highScores[t][j] % fuzz)));
		}
		DataPoint d(t, 0); //Create a data poin
		pair<string, DataPoint> point (hash, d); //Pair it with the hash we calculated
		storage.insert(point);
	}

	//Housekeeping
	delete fftData;
	delete highScores;

	return storage;
}

long powerOfTwo(long input)
{
	int twos[] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 
		1024, 2048, 4096, 8192, 16384, 32768, 65536, 
		131072, 262144, 524288, 1048576, 2097152, 4194304,
		8388608, 16777216, 33554432, 67108864, 134217728, 268435456,
		536870912, 1073741824};

	int i = 0;

	while (twos[i] < input)
	{
		i++;
	}

	return twos[i];
}

//This doesn't do it's job right.
int getIndex(int freq)
{
	int RANGE[] = {40, 80, 120, 180, UPPER_LIMIT+1};
	int i = 0; 
	while(RANGE[i] < freq) 
	{
		i++;
	}
	return i;
};

//long* readData(long* sampleLength)
//{
//	//Open the File
//	FILE *f = fopen("test.wav", "rb");
//	
//	long dataPointer = 40;
//
//	fseek(f, dataPointer, SEEK_SET); // Move the File pointer to data subchunk
//
//	//Read the size from the subchunk header
//	LongFromChar val;
//	byte a = fgetc(f);
//	byte b = fgetc(f);
//	byte c = fgetc(f);
//	byte d = fgetc(f);
//
//	long size = charToLong(a,b,c,d);
//	*sampleLength = size;
//	int index = 0;
//
//	int difference = powerOfTwo(size) - size;
//	long* wavData = new long[size + difference];
//
//   /*The data subchunk is arranged with interleaved channels
//	* [channel0][channel1][channel0][channel1]
//	*  short	 short	   short	 short
//	*/
//	while (dataPointer < size + 40)
//	{
//		a = fgetc(f);
//		b = fgetc(f);
//		c = fgetc(f);
//		d = fgetc(f);
//		wavData[index] = charToShort(a,b);
//		dataPointer += 4; //Skip to the next block
//		index ++;
//	}
//	while (dataPointer < (size + difference - 1))
//	{
//		wavData[index] = 0;
//		dataPointer ++;
//		index++;
//	}
//
//	fclose(f);
//	return wavData;
//}