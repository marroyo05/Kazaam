#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <hipfft/hipfft.h>
#include "byteStructure.hpp"

#include <cmath>
#include <iostream>
#include <fstream>
#include <vector>

using namespace std;
hipfftReal* readData(long* sampleLength);
int getIndex(int freq);

int main()
{
	int UPPER_LIMIT = 0;
	int LOWER_LIMIT = -1;

	long sampleLength; //Read the length of the sample
	int deltaT = 1; //timescale
	hipError_t cudaStatus;

	//Host Memory
	hipfftReal *wavData = readData(&sampleLength);
	hipfftComplex *fftData = new hipfftComplex[sampleLength];

	//Device Memory
	hipfftReal *d_wavData;
	hipfftComplex *d_fftData;


	cudaStatus = hipMalloc(&d_wavData, sampleLength * sizeof(hipfftReal));
	 if (cudaStatus != hipSuccess) {
        fprintf(stderr, "d_wav malloc failed");
    }

	 cudaStatus = hipMalloc(&d_fftData, (sampleLength) * sizeof(hipfftComplex));
	 if (cudaStatus != hipSuccess) {
        fprintf(stderr, "d_fft malloc failed");
		return cudaStatus;
    }

	//Copy over wav data
	 cudaStatus = hipMemcpy(d_wavData, wavData, sampleLength * sizeof(hipfftReal), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "d_wavData Memcpy failed");
		return cudaStatus;
    }

	//The actual transform
	hipfftHandle plan;
	hipfftPlan1d(&plan, sampleLength, HIPFFT_R2C, 1);
	hipfftExecR2C(plan, (hipfftReal* )d_wavData, (hipfftComplex*) d_fftData);

	//Copy fft data back
	cudaStatus = hipMemcpy(fftData, d_fftData, sampleLength * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "fftData Memcpy failed\n");
		return cudaStatus;
    }

	vector<float> magVector;
	//Spectral Reduction -- add a dimension???
	int freq = 1;
	for (int i = 0; i < sampleLength; i++)
	{
		magVector.push_back(log(fftData[i].x));
		freq += (int) (log10(i) * log10(i));
	}

	//Grab highest magnitudes
	for (int freq = LOWER_LIMIT; freq < UPPER_LIMIT-1; freq++)
	{
		//magnitude.get()
		double magnitude = log(abs(fftData[freq].x + 1));

		//????
		int index = getIndex(freq);

		//we only want the highest magnitude
		if (magnitude > highscores[index]) //high scores needs to be float[5]
		{
			highscores[index] = magnitude;
			recordPoints[index] = freq;
		}

	}
	//Hashing


	//Housekeeping
	delete wavData;
	delete fftData;

	hipFree(d_wavData);
	hipFree(d_fftData);

	hipfftDestroy(plan);
	hipDeviceReset();

	system("PAUSE");
	return 0;
}


int getIndex(int freq)
{
	int RANGE[] = {40, 80, 120, 180, UPPER_LIMIT+1};
	int i = 0; 
	while(RANGE[i] < freq) 
	{
		i++;
	}
	return i;
};

hipfftReal* readData(long* sampleLength)
{
	//Open the File
	FILE *f = fopen("test.wav", "rb");
	
	long dataPointer = 40;

	fseek(f, dataPointer, SEEK_SET); // Move the File pointer to data subchunk

	//Read the size from the subchunk header
	LongFromChar val;
	byte a = fgetc(f);
	byte b = fgetc(f);
	byte c = fgetc(f);
	byte d = fgetc(f);

	long size = charToLong(a,b,c,d);
	*sampleLength = size;
	int index = 0;

	hipfftReal* wavData = new hipfftReal[size];
   /*The data subchunk is arranged with interleaved channels
	* [channel0][channel1][channel0][channel1]
	*  short	 short	   short	 short
	*/
	while (dataPointer < size + 40)
	{
		a = fgetc(f);
		b = fgetc(f);
		c = fgetc(f);
		d = fgetc(f);
		wavData[index] = charToShort(a,b);
		dataPointer += 4; //Skip to the next block
		index ++;
	}

	fclose(f);
	dataPointer = 40; // Reset data pointer
	return wavData;
}