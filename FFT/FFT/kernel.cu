#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <hipfft/hipfft.h>
#include "byteStructure.hpp"

#include <cmath>
#include <iostream>
#include <fstream>
#include <vector>

#define LOWER_LIMIT 40
#define UPPER_LIMIT 300

#define CHUNK_SIZE 2048

using namespace std;
long* readData(long* sampleLength);
long powerOfTwo(long input);
int getIndex(int freq);


int main()
{
	long sampleLength; //Read the length of the sample
	hipError_t cudaStatus;

	//Load wav file into memory
	long *audio = readData(&sampleLength);
	int numChunks = sampleLength / CHUNK_SIZE;

	//Host memory
	hipfftComplex **wavData = new hipfftComplex* [numChunks];
	hipfftComplex **fftData = new hipfftComplex* [numChunks];

	//Creating Complex numbers from our readings
	for (int times = 0; times < numChunks; times++)
	{
		hipfftComplex *complex = new hipfftComplex[CHUNK_SIZE];
		fftData[times] = new hipfftComplex[CHUNK_SIZE];
		for (int i = 0; i < CHUNK_SIZE; i++)
		{
			complex[i].x = audio[times * CHUNK_SIZE + i];
			complex[i].y = 0;
		}
		wavData[times] = complex;
	}

	hipfftComplex *d_wavData;
	hipfftComplex *d_fftData;

	//The actual transform
	hipfftHandle plan;
	hipfftPlan1d(&plan, CHUNK_SIZE, HIPFFT_C2C, 1);

	//FFT
	for (int i = 0; i < numChunks; i++)
	{
		cudaStatus = hipMalloc(&d_wavData, CHUNK_SIZE  * sizeof(hipfftComplex));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "d_wav malloc failed");
			return cudaStatus;
		}

		cudaStatus = hipMalloc(&d_fftData, CHUNK_SIZE * sizeof(hipfftComplex));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "d_fft malloc failed");
			return cudaStatus;
		}

		//Copy over wav data
		cudaStatus = hipMemcpy(d_wavData, wavData[i], CHUNK_SIZE * sizeof(hipfftComplex), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "d_wavData Memcpy failed");
			return cudaStatus;
		}

		hipfftExecC2C(plan, (hipfftComplex*)d_wavData, (hipfftComplex*) d_fftData, 1);

		//Copy fft data back
		cudaStatus = hipMemcpy(fftData[i], d_fftData, CHUNK_SIZE * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "fftData Memcpy failed\n");
			system("PAUSE");
			return cudaStatus;
		}
	}
	
	delete wavData;
	hipFree(d_wavData);
	hipFree(d_fftData);
	hipfftDestroy(plan);
	hipDeviceReset();

	cout << "FFT Complete." << endl;

	double **highScores = new double * [numChunks];
	for (int i = 0; i < numChunks; i ++)
	{
		highScores[i] = new double[5];
		for (int j = 0; j < 5; j++)
		{
			highScores[i][j] = 0;
		}
		
	}

	//Scale the data down
	for (int t = 0; t < numChunks; t++)
	{
		for (int freq = LOWER_LIMIT; freq < UPPER_LIMIT - 1; freq ++)
		{
			double magnitude = log(abs(fftData[t][freq].x)) + 1;

			int index = getIndex(freq);

			if (magnitude > highScores[t][index])
			{
				highScores[t][index] = magnitude;
			}
		}
	}

	cout << "\a" << endl;
	//Display to test things
	for (int i = 0; i < 5; i++)
	{
		for (int j = 0; j < 5; j++)
		{
			cout << highScores[i][j] << " ";
		}
		cout << endl;
	}
	
	//Housekeeping
	delete fftData;

	system("PAUSE");
	return 0;
}

long powerOfTwo(long input)
{
	int twos[] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 
		1024, 2048, 4096, 8192, 16384, 32768, 65536, 
		131072, 262144, 524288, 1048576, 2097152, 4194304,
		8388608, 16777216, 33554432, 67108864, 134217728, 268435456,
		536870912, 1073741824};

	int i = 0;

	while (twos[i] < input)
	{
		i++;
	}

	return twos[i];
}

int getIndex(int freq)
{
	int RANGE[] = {40, 80, 120, 180, UPPER_LIMIT+1};
	int i = 0; 
	while(RANGE[i] < freq) 
	{
		i++;
	}
	return i;
};

long* readData(long* sampleLength)
{
	//Open the File
	FILE *f = fopen("test.wav", "rb");
	
	long dataPointer = 40;

	fseek(f, dataPointer, SEEK_SET); // Move the File pointer to data subchunk

	//Read the size from the subchunk header
	LongFromChar val;
	byte a = fgetc(f);
	byte b = fgetc(f);
	byte c = fgetc(f);
	byte d = fgetc(f);

	long size = charToLong(a,b,c,d);
	*sampleLength = size;
	int index = 0;

	int difference = powerOfTwo(size) - size;
	long* wavData = new long[size + difference];

   /*The data subchunk is arranged with interleaved channels
	* [channel0][channel1][channel0][channel1]
	*  short	 short	   short	 short
	*/
	while (dataPointer < size + 40)
	{
		a = fgetc(f);
		b = fgetc(f);
		c = fgetc(f);
		d = fgetc(f);
		wavData[index] = charToShort(a,b);
		dataPointer += 4; //Skip to the next block
		index ++;
	}
	while (dataPointer < (size + difference - 1))
	{
		wavData[index] = 0;
		dataPointer ++;
		index++;
	}

	fclose(f);
	return wavData;
}